#include "hip/hip_runtime.h"
#include "cuda_header.cuh"
#include "open_acc_map_h.cuh"
#include "../vlasovsolver/vec.h"
#include "../definitions.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NPP_MAXABS_32F ( 3.402823466e+38f )
#define NPP_MINABS_32F ( 1.175494351e-38f )

#define NPP_MAXABS_64F ( 1.7976931348623158e+308 )
#define NPP_MINABS_64F ( 2.2250738585072014e-308 )

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ));

__constant__ int acc_semilag_flag;
//__constant__ int WID_DEVICE = WID;

/*
#if VECTORCLASS_H >= 20000
  __constant__ int VECTORCLASS_H_DEVICE = 20102;
#else
  __constant__ int VECTORCLASS_H_DEVICE = 0;
#endif
*/

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ));
static void HandleError( hipError_t err, const char *file, int line )
{
    if (err != hipSuccess)
    {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ), file, line );
        exit( EXIT_FAILURE );
    }
}

__device__ Vec minmod(const Vec slope1, const Vec slope2)
{
  Vec zero(0.0);
  Vec slope = select(abs(slope1) < abs(slope2), slope1, slope2);
  return select(slope1 * slope2 <= 0, zero, slope);
}
__device__ Vec maxmod(const Vec slope1, const Vec slope2)
{
  Vec zero(0.0);
  Vec slope=select(abs(slope1) > abs(slope2), slope1, slope2);
  //check for extrema
  return select(slope1 * slope2 <= 0, zero, slope);
}
__device__ Vec slope_limiter_sb(Vec& l, Vec& m, Vec& r)
{
  Vec a = r-m;
  Vec b = m-l;
  Vec slope1 = minmod(a, 2*b);
  Vec slope2 = minmod(2*a, b);
  return maxmod(slope1, slope2);
}
__device__ Vec slope_limiter(Vec& l,Vec& m,Vec& r)
{
   return slope_limiter_sb(l,m,r);
}
__device__ void compute_plm_coeff(Vec *values, uint k, Vec *a, Realv threshold)
{
  // scale values closer to 1 for more accurate slope limiter calculation
  Realv scale = 1./threshold;
  Vec v_1 = values[k - 1] * scale;
  Vec v_2 = values[k] * scale;
  Vec v_3 = values[k + 1] * scale;
  Vec d_cv = slope_limiter(v_1, v_2, v_3) * threshold;
  a[0] = values[k] - d_cv * 0.5;
  a[1] = d_cv * 0.5;
}

__device__ int i_pcolumnv(int j, int k, int k_block, int num_k_blocks, int WID_device)
{
  return ((j) / ( VECL / WID_device)) * WID_device * ( num_k_blocks + 2) + (k) + ( k_block + 1 ) * WID_device;
}

__global__ void acceleration_1
(
  Realf *dev_blockData,
  int totalColumns,
  Column *dev_columns,
  Vec *values,
  int *dev_cell_indices_to_id,
  int WID_device,
  Realv intersection,
  Realv intersection_di,
  Realv intersection_dj,
  Realv intersection_dk,
  Realv minValue,
  Realv dv,
  Realv v_min
)
{
  //printf("CUDA 1\n");
  for( uint column=0; column < totalColumns; column++)
  {
    //printf("CUDA 2\n");
     // i,j,k are relative to the order in which we copied data to the values array.
     // After this point in the k,j,i loops there should be no branches based on dimensions
     // Note that the i dimension is vectorized, and thus there are no loops over i
     // Iterate through the perpendicular directions of the column
     for (uint j = 0; j < WID_device; j += VECL/WID_device)
     {
       //printf("CUDA 3\n");
        const vmesh::LocalID nblocks = dev_columns[column].nblocks;
        // create vectors with the i and j indices in the vector position on the plane.
        #if VECL == 4
          const Veci i_indices = Veci(0, 1, 2, 3);
          const Veci j_indices = Veci(j, j, j, j);
        #elif VECL == 8
          const Veci i_indices = Veci(0, 1, 2, 3, 0, 1, 2, 3);
          const Veci j_indices = Veci(j, j, j, j, j + 1, j + 1, j + 1, j + 1);
        #elif VECL == 16
          const Veci i_indices = Veci(0, 1, 2, 3,
                                      0, 1, 2, 3,
                                      0, 1, 2, 3,
                                      0, 1, 2, 3);
          const Veci j_indices = Veci(j, j, j, j,
                                      j + 1, j + 1, j + 1, j + 1,
                                      j + 2, j + 2, j + 2, j + 2,
                                      j + 3, j + 3, j + 3, j + 3);
        #endif
        const Veci  target_cell_index_common =
           i_indices * dev_cell_indices_to_id[0] +
           j_indices * dev_cell_indices_to_id[1];

        // intersection_min is the intersection z coordinate (z after
        // swaps that is) of the lowest possible z plane for each i,j
        // index (i in vector)
        const Vec intersection_min =
           intersection +
           (dev_columns[column].i * WID_device + to_realv(i_indices)) * intersection_di +
           (dev_columns[column].j * WID_device + to_realv(j_indices)) * intersection_dj;

        /*compute some initial values, that are used to set up the
         * shifting of values as we go through all blocks in
         * order. See comments where they are shifted for
         * explanations of their meaning*/
        Vec v_r0( (WID_device * dev_columns[column].kBegin) * dv + v_min);
        Vec lagrangian_v_r0((v_r0-intersection_min)/intersection_dk);

        /* compute location of min and max, this does not change for one
        column (or even for this set of intersections, and can be used
        to quickly compute max and min later on*/
        //TODO, these can be computed much earlier, since they are
        //identiacal for each set of intersections
        int minGkIndex=0, maxGkIndex=0; // 0 for compiler
        {
            #if defined (VEC4D_FALLBACK) || defined (VEC8D_FALLBACK)
            Realv maxV = NPP_MAXABS_64F;
            Realv minV = NPP_MINABS_64F;
            #endif
            #if defined (VEC4F_FALLBACK) || defined (VEC8F_FALLBACK)
            Realv maxV = NPP_MAXABS_32F;
            Realv minV = NPP_MINABS_32F;
            #endif
           //Realv maxV = std::numeric_limits<Realv>::min();
           //Realv minV = std::numeric_limits<Realv>::max();
           for(int i = 0; i < VECL; i++)
           {
              if ( lagrangian_v_r0[i] > maxV)
              {
                 maxV = lagrangian_v_r0[i];
                 maxGkIndex = i;
              }
              if ( lagrangian_v_r0[i] < minV)
              {
                 minV = lagrangian_v_r0[i];
                 minGkIndex = i;
              }
           }
        }
        // loop through all blocks in column and compute the mapping as integrals.
        for (uint k=0; k < WID_device * nblocks; ++k )
        {
          //printf("CUDA 4\n");
           // Compute reconstructions
           // values + i_pcolumnv(n_cblocks, -1, j, 0) is the starting point of the column data for fixed j
           // k + WID is the index where we have stored k index, WID amount of padding.
            //if(acc_semilag_flag==0)
            //{
              //Vec a[2];
              Vec *a = new Vec[2];
              compute_plm_coeff(values + dev_columns[column].valuesOffset + i_pcolumnv(j, 0, -1, nblocks, WID_device), k + WID_device, a, minValue);
            //}
            /*
            if(acc_semilag_flag==1)
            {
              Vec a[3];
              compute_ppm_coeff(values + columns[column].valuesOffset  + i_pcolumnv(j, 0, -1, nblocks), h4, k + WID, a, minValue);
            }
            if(acc_semilag_flag==2)
            {
              Vec a[5];
              compute_pqm_coeff(values + columns[column].valuesOffset  + i_pcolumnv(j, 0, -1, nblocks), h8, k + WID, a, minValue);
            }
            */
           // set the initial value for the integrand at the boundary at v = 0
           // (in reduced cell units), this will be shifted to target_density_1, see below.
           Vec target_density_r(0.0);
           // v_l, v_r are the left and right velocity coordinates of source cell.
           Vec v_r = v_r0  + (k+1)* dv;
           Vec v_l = v_r0  + k* dv;
           // left(l) and right(r) k values (global index) in the target
           // Lagrangian grid, the intersecting cells. Again old right is new left.
           Veci lagrangian_gk_l,lagrangian_gk_r;
           /*
           if(VECTORCLASS_H_DEVICE >= 20000)
            {
              lagrangian_gk_r = truncatei((v_l-intersection_min)/intersection_dk);
              lagrangian_gk_r = truncatei((v_r-intersection_min)/intersection_dk);
            }
            else
            {
              lagrangian_gk_l = truncate_to_int((v_l-intersection_min)/intersection_dk);
              lagrangian_gk_r = truncate_to_int((v_r-intersection_min)/intersection_dk);
            }
            */
            // I keep only this version with Fallback, because the version with Agner requires another call to CPU
            lagrangian_gk_l = truncate_to_int((v_l-intersection_min)/intersection_dk);
            lagrangian_gk_r = truncate_to_int((v_r-intersection_min)/intersection_dk);
           //limits in lagrangian k for target column. Also take into
           //account limits of target column
           int minGk = max(int(lagrangian_gk_l[minGkIndex]), int(dev_columns[column].minBlockK * WID_device));
           int maxGk = min(int(lagrangian_gk_r[maxGkIndex]), int((dev_columns[column].maxBlockK + 1) * WID_device - 1));
           // Run along the column and perform the polynomial reconstruction
           //for(int gk = minGk; gk <= maxGk; gk++){
           for(int gk = dev_columns[column].minBlockK * WID_device; gk <= dev_columns[column].maxBlockK * WID_device; gk++)
           {
             //printf("CUDA 5\n");
              if(gk < minGk || gk > maxGk)
              {
                 continue;
              }
              const int blockK = gk/WID_device;
              const int gk_mod_WID = (gk - blockK * VECL);
              //the block of the Lagrangian cell to which we map
              //const int target_block(target_block_index_common + blockK * block_indices_to_id[2]);
              //cell indices in the target block  (TODO: to be replaced by
              //compile time generated scatter write operation)
              const Veci target_cell(target_cell_index_common + gk_mod_WID * dev_cell_indices_to_id[2]);
              //the velocity between which we will integrate to put mass
              //in the targe cell. If both v_r and v_l are in same cell
              //then v_1,v_2 should be between v_l and v_r.
              //v_1 and v_2 normalized to be between 0 and 1 in the cell.
              //For vector elements where gk is already larger than needed (lagrangian_gk_r), v_2=v_1=v_r and thus the value is zero.
              const Vec v_norm_r = (  min(  max( (gk + 1) * intersection_dk + intersection_min, v_l), v_r) - v_l) * (1.0/dv);
              /*shift, old right is new left*/
              const Vec target_density_l = target_density_r;
              // compute right integrand
              if(acc_semilag_flag==0)
                target_density_r = v_norm_r * ( a[0] + v_norm_r * a[1] );
              if(acc_semilag_flag==1)
                target_density_r = v_norm_r * ( a[0] + v_norm_r * ( a[1] + v_norm_r * a[2] ) );
              if(acc_semilag_flag==2)
                target_density_r =
                  v_norm_r * ( a[0] + v_norm_r * ( a[1] + v_norm_r * ( a[2] + v_norm_r * ( a[3] + v_norm_r * a[4] ) ) ) );
              //store values, one element at a time. All blocks have been created by now.
              //TODO replace by vector version & scatter & gather operation
              const Vec target_density = target_density_r - target_density_l;
              for (int target_i=0; target_i < VECL; ++target_i)
              {
                // do the conversion from Realv to Realf here, faster than doing it in accumulation
                const Realf tval = target_density[target_i];
                const uint tcell = target_cell[target_i];
                (&dev_blockData[dev_columns[column].targetBlockOffsets[blockK]])[tcell] += tval;
              }  // for-loop over vector elements
           } // for loop over target k-indices of current source block
        } // for-loop over source blocks
     } //for loop over j index
  } //for loop over columns
}

Realf* acceleration_1_wrapper
(
  int bdsw3,
  Realf *blockData,
  int totalColumns,
  Column *columns,
  int valuesSizeRequired,
  Vec *values,
  uint cell_indices_to_id[],
  Realv intersection,
  Realv intersection_di,
  Realv intersection_dj,
  Realv intersection_dk,
  Realv v_min,
  Realv dv,
  Realv minValue
)
{
  printf("STAGE 3\n");

//    hipMemcpyToSymbol(HIP_SYMBOL("WID_DEVICE"), &WID_DEVICE, sizeof(int));
//  hipMemcpyToSymbol(HIP_SYMBOL("VECTORCLASS_H_DEVICE"), &VECTORCLASS_H_DEVICE, sizeof(int));

  int acc_semilag_flag = 0;
  #ifdef ACC_SEMILAG_PLM
    acc_semilag_flag = 0;
  #endif
  #ifdef ACC_SEMILAG_PPM
    acc_semilag_flag = 1;
  #endif
  #ifdef ACC_SEMILAG_PQM
    acc_semilag_flag = 2;
  #endif
  hipMemcpyToSymbol(HIP_SYMBOL("acc_semilag_flag"), &acc_semilag_flag, sizeof(int));

  Realf *dev_blockData;
  HANDLE_ERROR( hipMalloc((void**)&dev_blockData, bdsw3*sizeof(Realf)));
  HANDLE_ERROR( hipMemcpy(dev_blockData, blockData, bdsw3*sizeof(Realf), hipMemcpyHostToDevice));

  Column *dev_columns;
  HANDLE_ERROR( hipMalloc((void**)&dev_columns, totalColumns*sizeof(Column)));
  HANDLE_ERROR( hipMemcpy(dev_columns, columns, totalColumns*sizeof(Column), hipMemcpyHostToDevice));

  Vec *dev_values;
  HANDLE_ERROR( hipMalloc((void**)&dev_values, valuesSizeRequired*sizeof(Vec)));
  HANDLE_ERROR( hipMemcpy(dev_values, values, valuesSizeRequired*sizeof(Vec), hipMemcpyHostToDevice));

  int *dev_cell_indices_to_id;
  HANDLE_ERROR( hipMalloc((void**)&dev_cell_indices_to_id, 3*sizeof(int)));
  HANDLE_ERROR( hipMemcpy(dev_cell_indices_to_id, cell_indices_to_id, 3*sizeof(int), hipMemcpyHostToDevice));

  int WID_device = WID;
  printf("WID_device = %d\n", WID_device);
  acceleration_1<<<BLOCKS, THREADS>>>
  (
    dev_blockData,
    totalColumns,
    dev_columns,
    dev_values,
    dev_cell_indices_to_id,
    WID_device,
    intersection,
    intersection_di,
    intersection_dj,
    intersection_dk,
    minValue,
    dv,
    v_min
  );

  HANDLE_ERROR( hipMemcpy(blockData, dev_blockData, bdsw3*sizeof(Realf), hipMemcpyDeviceToHost));

  HANDLE_ERROR( hipFree(dev_blockData) );
  HANDLE_ERROR( hipFree(dev_cell_indices_to_id) );
  HANDLE_ERROR( hipFree(dev_columns) );
  HANDLE_ERROR( hipFree(dev_values) );

  return blockData;
}
